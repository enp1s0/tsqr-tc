#include "hip/hip_runtime.h"
#include <cstdint>
#include <mma.h>
#include <cutf/type.hpp>
#include <cutf/math.hpp>
#include <cutf/thread.hpp>
#include <wmma_extension.hpp>

#include <tsqr_tc/batchedqr.hpp>

namespace {

constexpr unsigned warp_size = 32u;

// This function fills memory with zero
template <unsigned block_size, unsigned size, class T>
__device__ void fill_zero(T* const ptr) {
	if constexpr (size % block_size == 0) {
		for (unsigned i = 0; i < size; i += block_size) {
			const auto index = i + threadIdx.x;
			ptr[index] = cutf::type::cast<T>(0.0f);
		}
	} else {
		for (unsigned index = threadIdx.x; index < size; index += block_size) {
			ptr[index] = cutf::type::cast<T>(0.0f);
		}
	}
}

// This function copies matrix data from global memory to shared memory
// Ristrictions:
// - smem_m == block_size
template <unsigned block_size, unsigned smem_n, unsigned smem_ld, class SMEM_T, class GMEM_T>
__device__ void copy_matrix_g2s(
		SMEM_T* const smem,
		const GMEM_T* const gmem_ptr, const std::size_t gmem_ld,
		const std::size_t m, const std::size_t n
		) {
	if (m == block_size) {
		unsigned i_n = 0;
		for (; i_n < n; i_n++) {
			const auto v = gmem_ptr[gmem_ld * i_n + threadIdx.x];
			smem[smem_ld * i_n + threadIdx.x] = cutf::type::cast<SMEM_T>(v);
		}
		for (; i_n < smem_n; i_n++) {
			smem[smem_ld * i_n + threadIdx.x] = cutf::type::cast<SMEM_T>(0.0f);
		}
	} else {
		unsigned i_n = 0;
		for (; i_n < n; i_n++) {
			const auto v = cutf::type::cast<GMEM_T>(0.0f);
			if (threadIdx.x < m) {
				v = gmem_ptr[gmem_ld * i_n + threadIdx.x];
			}
			smem[smem_ld * i_n + threadIdx.x] = cutf::type::cast<SMEM_T>(v);
		}
		for (; i_n < smem_n; i_n++) {
			smem[smem_ld * i_n + threadIdx.x] = cutf::type::cast<SMEM_T>(0.0f);
		}
	}
}

// This function copies matrix data from shared memory to global memory
// Ristrictions:
// - smem_m == block_size
template <unsigned block_size, unsigned smem_n, unsigned smem_ld, class SMEM_T, class GMEM_T>
__device__ void copy_matrix_s2g(
		GMEM_T* const gmem_ptr, const std::size_t gmem_ld,
		const SMEM_T* const smem,
		const std::size_t m, const std::size_t n
		) {
	if (m == block_size) {
		unsigned i_n = 0;
		for (; i_n < n; i_n++) {
			const auto v = smem[smem_ld * i_n + threadIdx.x];
			gmem_ptr[gmem_ld * i_n + threadIdx.x] = cutf::type::cast<GMEM_T>(v);
		}
	} else {
		if (threadIdx.x < m) {
			for (unsigned i_n = 0; i_n < n; i_n++) {
				const auto v = smem[smem_ld * i_n + threadIdx.x];
				gmem_ptr[gmem_ld * i_n + threadIdx.x] = cutf::type::cast<GMEM_T>(v);
			}
		}
	}
}

// This function computes L2-norm ^2 of a given vector(array).
// Restrictions:
// - size % warp_size == 0
template <class COMPUTE_T, class T>
__device__ COMPUTE_T compute_norm2(const T* const ptr, const unsigned size) {
	auto norm2 = cutf::type::cast<COMPUTE_T>(0.0f);
	for (unsigned i = 0; i < size; i += warp_size) {
		const auto v = cutf::type::cast<COMPUTE_T>(ptr[i + cutf::thread::get_lane_id()]);
		norm2 += v * v;
	}
	for(auto mask = (warp_size >> 1); mask > 0; mask >>= 1) {
		norm2 += __shfl_xor_sync(0xffffffff, norm2, mask);
	}
	return norm2;
}

// This function accumulates vectors on shared memory.
// Restrictions:
// count == block_size / war_size
// output_ptr == inpute_ptr
template <unsigned block_size, class T>
__device__ void accumulate_vectors(T* const smem_vec_ptr, const unsigned vec_len) {
	for (unsigned whole_vec_len = vec_len * block_size / warp_size; whole_vec_len > vec_len; whole_vec_len >>= 1) {
		for (unsigned offset = 0; offset < whole_vec_len / 2; offset += block_size) {
			const auto index = offset + threadIdx.x;
			if (index > vec_len) break;

			smem_vec_ptr[index] += smem_vec_ptr[index + whole_vec_len / 2];
		}
		__syncthreads();
	}
}

// This function computes `tmp = y^t * A`.
// Restrictions:
// - smem_m == block_size
// - smem_n == DIM_BLOCK_N
template <unsigned smem_m, unsigned smem_n, unsigned smem_ldm>
__device__ void compute_reflection_0_fp32_hmma_cor(
		float* const smem_reduction,
		const float* const smem_y,
		const float* const smem_A
		) {
	constexpr unsigned num_accumulate = warp_size / smem_n;
	constexpr float cor_scale = 1024.0f;
	nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, smem_n, smem_n, smem_n, half, nvcuda::wmma::row_major> frag_yt[num_accumulate], frag_d_yt[num_accumulate];
	nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, smem_n, smem_n, smem_n, half, nvcuda::wmma::col_major> frag_a[num_accumulate], frag_d_a[num_accumulate];
	nvcuda::wmma::fragment<nvcuda::wmma::accumulator, smem_n, smem_n, smem_n, float> frag_ytA, frag_d_ytA;
	mtk::wmma::fill_zero(frag_ytA);
	mtk::wmma::fill_zero(frag_d_ytA);

	// Load A
	mtk::wmma::foreach<decltype(frag_a[0])>([&](const unsigned frag_index_list[], const unsigned frag_index_count, const unsigned mem_index) {
				const auto offset = (mem_index / smem_n) * smem_ldm;
				const auto row = mem_index % smem_n + (threadIdx.x & 0xffffffe0u);
				for (unsigned k = 0; k < num_accumulate; k++) {
					const auto r = row + k * smem_n;
					const auto v = smem_A[offset + r];
					const auto hv = cutf::type::cast<half>(v);
					const auto dhv = cutf::type::cast<half>((v - cutf::type::cast<float>(hv)) * cor_scale);
					for (unsigned i = 0; i < frag_index_count; i++) {
					const unsigned frag_index = frag_index_list[i];
						frag_a[k].x[frag_index] = hv;
						frag_d_a[k].x[frag_index] = dhv;
					}
				}
			});
	mtk::wmma::foreach_v<decltype(frag_yt[0])>(
			[&](const unsigned* frag_index_list, const unsigned fragment_index_count, const unsigned mem_index) {
				for (unsigned k = 0; k < num_accumulate; k++) {
					const auto row = k * smem_n + mem_index + (threadIdx.x & 0xffffffe0u);
					const auto v = smem_y[row];
					const auto hv = cutf::type::cast<half>(v);
					const auto dhv = cutf::type::cast<half>((v - cutf::type::cast<float>(hv)) * cor_scale);
					for (unsigned i = 0; i < fragment_index_count; i++) {
						const auto frag_index = frag_index_list[i];
						frag_yt[k].x[frag_index] = hv;
						frag_d_yt[k].x[frag_index] = dhv;
					}
				}
			});

	for (unsigned k = 0; k < num_accumulate; k++) {
		// Compute (y^t * A)
		nvcuda::wmma::mma_sync(frag_ytA  , frag_a[k]  , frag_yt[k], frag_ytA  );
		nvcuda::wmma::mma_sync(frag_d_ytA, frag_d_a[k], frag_yt[k], frag_d_ytA);
		nvcuda::wmma::mma_sync(frag_d_ytA, frag_a[k], frag_d_yt[k], frag_d_ytA);
	}

	// Store
	mtk::wmma::foreach_v<decltype(frag_ytA)>(nvcuda::wmma::mem_row_major,
			[&](const unsigned* frag_index_list, const unsigned fragment_index_count, const unsigned mem_index) {
				float* res_ptr = smem_reduction + smem_n * (threadIdx.x >> 5);
				for (unsigned i = 0; i < fragment_index_count; i++) {
					const auto frag_index = frag_index_list[i];
					res_ptr[mem_index] = frag_ytA.x[frag_index] + frag_d_ytA.x[frag_index] / cor_scale;
				}
			});

	// Accumulate
	__syncthreads();
	accumulate_vectors<smem_m>(smem_reduction, smem_n);
}

template <mtk::tsqr_tc::compute_mode::type compute_mode, unsigned smem_m, unsigned smem_n, unsigned smem_ldm>
__device__ void compute_reflection_0(
		typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const smem_reduction_ptr,
		const typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const smem_y_ptr,
		const typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const smem_a_ptr
		) {
	if constexpr (compute_mode == mtk::tsqr_tc::compute_mode::fp32_hmma_cor) {
		compute_reflection_0_fp32_hmma_cor<smem_m, smem_n, smem_ldm>(smem_reduction_ptr, smem_y_ptr, smem_a_ptr);
	}
}

// This function computes `A = A -2t * y * tmp`.
// Restrictions:
// - smem_m == block_size
// - smem_n == DIM_BLOCK_N
template <unsigned smem_m, unsigned smem_n, unsigned smem_ldm>
__device__ void compute_reflection_1_fp32_hmma_cor(
		float* const smem_A_ptr,
		float* const smem_reduction_ptr,
		const float* const smem_y_ptr,
		const float t
		) {
	constexpr unsigned num_col_block = warp_size / smem_n;

	if (threadIdx.x < smem_n) {
		smem_reduction_ptr[threadIdx.x] *= -t;
	}

	nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, smem_n, smem_n, smem_n, half, nvcuda::wmma::col_major> frag_y;
	nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, smem_n, smem_n, smem_n, half, nvcuda::wmma::row_major> frag_tmp;
	nvcuda::wmma::fragment<nvcuda::wmma::accumulator, smem_n, smem_n, smem_n, float> frag_A;

	mtk::wmma::fill_fragment(frag_y);
	__syncthreads();
	mtk::wmma::make_direct_product_fragment(frag_tmp, smem_reduction_ptr);

	for (unsigned i = 0; i < num_col_block; i++) {
		mtk::wmma::make_direct_product_fragment(frag_y, smem_y_ptr + i * smem_n + (threadIdx.x & 0xffffffe0u));
		nvcuda::wmma::load_matrix_sync(frag_A, smem_A_ptr	 + i * smem_n + (threadIdx.x & 0xffffffe0u), smem_ldm);

		nvcuda::wmma::mma_sync(frag_A, frag_tmp, frag_y, frag_A);

		nvcuda::wmma::store_matrix_sync(smem_A_ptr + i * smem_n + (threadIdx.x & 0xffffffe0u), frag_A, smem_ldm);
	}
}

template <mtk::tsqr_tc::compute_mode::type compute_mode, unsigned smem_m, unsigned smem_n, unsigned smem_ldm>
__device__ void compute_reflection_1(
		typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const smem_A_ptr,
		const typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const smem_reduction_ptr,
		const typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const smem_y_ptr,
		const typename mtk::tsqr_tc::detail::get_type<compute_mode>::type t
		) {
	if constexpr (compute_mode == mtk::tsqr_tc::compute_mode::fp32_hmma_cor) {
		compute_reflection_1_fp32_hmma_cor<smem_m, smem_n, smem_ldm>(smem_A_ptr, smem_reduction_ptr, smem_y_ptr, t);
	}
}

// This function computes `A = A -2t * y * tmp`.
// Restrictions:
// - smem_m == block_size
// - smem_n == DIM_BLOCK_N
template <unsigned smem_m, unsigned smem_n, unsigned smem_ldm>
__device__ void compute_w_fp32_hmma_cor(
		float* const smem_w_ptr,
		float* const smem_reduction_ptr,
		const float* const smem_y_ptr,
		const float* const smem_Y_ptr,
		const float* const smem_W_ptr,
		const float t
		) {
	constexpr unsigned num_col_block = warp_size / smem_n;
	const float cor_scale = 1024.0f;

	nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, smem_n, smem_n, smem_n, half, nvcuda::wmma::col_major> frag_y, frag_d_y;
	mtk::wmma::fill_zero(frag_y);
	mtk::wmma::fill_zero(frag_d_y);

	{
		nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, smem_n, smem_n, smem_n, half, nvcuda::wmma::row_major> frag_Yt[num_col_block], frag_d_Yt[num_col_block];
		nvcuda::wmma::fragment<nvcuda::wmma::accumulator, smem_n, smem_n, smem_n, float> frag_tmp, frag_d_tmp;
		mtk::wmma::fill_zero(frag_tmp);
		mtk::wmma::fill_zero(frag_d_tmp);

		// Load Yt
		mtk::wmma::foreach<decltype(frag_Yt[0])>([&](const unsigned frag_index_list[], const unsigned frag_index_count, const unsigned mem_index) {
					const auto offset = (mem_index / smem_n) * smem_ldm;
					const auto row = mem_index % smem_n + (threadIdx.x & 0xffffffe0u);
					for (unsigned k = 0; k < num_col_block; k++) {
						const auto r = row + k * smem_n;
						const auto v = smem_Y_ptr[offset + r];
						const auto hv = cutf::type::cast<half>(v);
						const auto dhv = cutf::type::cast<half>((v - cutf::type::cast<float>(hv)) * cor_scale);
						for (unsigned i = 0; i < frag_index_count; i++) {
							const unsigned frag_index = frag_index_list[i];
							frag_Yt[k].x[frag_index] = hv;
							frag_d_Yt[k].x[frag_index] = dhv;
						}
					}
				});
		// Load y

		for (unsigned k = 0; k < num_col_block; k++) {
			// Compute (Yt * y)
			nvcuda::wmma::mma_sync(frag_tmp  , frag_Yt[k]  , frag_y[k], frag_tmp  );
			nvcuda::wmma::mma_sync(frag_d_tmp, frag_d_Yt[k], frag_y[k], frag_d_tmp);
			nvcuda::wmma::mma_sync(frag_d_tmp, frag_Yt[k], frag_d_y[k], frag_d_tmp);
		}

		// Store
		mtk::wmma::foreach_v<decltype(frag_tmp)>(nvcuda::wmma::mem_row_major,
				[&](const unsigned* frag_index_list, const unsigned fragment_index_count, const unsigned mem_index) {
					float* res_ptr = smem_reduction_ptr + smem_n * (threadIdx.x >> 5);
					for (unsigned i = 0; i < fragment_index_count; i++) {
						const auto frag_index = frag_index_list[i];
						res_ptr[mem_index] = frag_tmp.x[frag_index] + frag_d_tmp.x[frag_index] / cor_scale;
					}
				});
	}

	// Accumulate
	__syncthreads();
	accumulate_vectors<smem_m>(smem_reduction_ptr, smem_n);

	// Compute w <- W * tmp
	{
		nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, smem_n, smem_n, smem_n, half, nvcuda::wmma::col_major> frag_W[num_col_block], frag_d_W[num_col_block];
		nvcuda::wmma::fragment<nvcuda::wmma::accumulator, smem_n, smem_n, smem_n, float> frag_w[num_col_block], frag_d_w[num_col_block];
		// Load tmp
		mtk::wmma::foreach_v<decltype(frag_w[0])>(
				[&](const unsigned* frag_index_list, const unsigned fragment_index_count, const unsigned mem_index) {
					const auto v = smem_reduction_ptr[mem_index];
					const auto hv = cutf::type::cast<half>(v);
					const auto dhv = cutf::type::cast<half>((v - cutf::type::cast<float>(hv)) * cor_scale);
					for (unsigned i = 0; i < fragment_index_count; i++) {
						const auto frag_index = frag_index_list[i];
						frag_y[0].x[frag_index] = hv;
						frag_d_y[0].x[frag_index] = dhv;
					}
				});
		// Load W
		mtk::wmma::foreach<decltype(frag_W[0])>([&](const unsigned frag_index_list[], const unsigned frag_index_count, const unsigned mem_index) {
					const auto offset = (mem_index / smem_n) * smem_ldm;
					const auto row = mem_index % smem_n + (threadIdx.x & 0xffffffe0u);
					for (unsigned k = 0; k < num_col_block; k++) {
						const auto r = row + k * smem_n;
						const auto v = smem_W_ptr[offset + r];
						const auto hv = cutf::type::cast<half>(v);
						const auto dhv = cutf::type::cast<half>((v - cutf::type::cast<float>(hv)) * cor_scale);
						for (unsigned i = 0; i < frag_index_count; i++) {
						const unsigned frag_index = frag_index_list[i];
							frag_W[k].x[frag_index] = hv;
							frag_d_W[k].x[frag_index] = dhv;
						}
					}
				});
		for (unsigned k = 0; k < num_col_block; k++) {
			mtk::wmma::fill_zero(frag_w);
			mtk::wmma::fill_zero(frag_d_w);
			// Compute (Yt * A)
			nvcuda::wmma::mma_sync(frag_w[k]  , frag_W[k]  , frag_y[k]  , frag_w  );
			nvcuda::wmma::mma_sync(frag_d_w[k], frag_d_W[k], frag_y[k]  , frag_d_w);
			nvcuda::wmma::mma_sync(frag_d_w[k], frag_W[k]  , frag_d_y[k], frag_d_w);
		}
		// Store
		mtk::wmma::foreach_v<decltype(frag_w[0])>(nvcuda::wmma::mem_row_major,
				[&](const unsigned* frag_index_list, const unsigned fragment_index_count, const unsigned mem_index) {
					for (unsigned k = 0; k < num_col_block; k++) {
						const auto offset = warp_size * (threadIdx.x >> 5) + k * smem_n;
						float* const res_ptr = smem_w_ptr + offset;
						const float* const y_ptr = smem_y_ptr + offset;
						for (unsigned i = 0; i < fragment_index_count; i++) {
							const auto frag_index = frag_index_list[i];
							res_ptr[mem_index] = (y_ptr[mem_index] - (frag_w[k].x[frag_index] + frag_d_w[k].x[frag_index] / cor_scale)) * t;
						}
					}
				});
	}
}

template <mtk::tsqr_tc::compute_mode::type compute_mode, unsigned smem_m, unsigned smem_n, unsigned smem_ldm>
__device__ void compute_w(
		typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const smem_w_ptr,
		const typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const smem_reduction_ptr,
		const typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const smem_y_ptr,
		const typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const smem_Y_ptr,
		const typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const smem_W_ptr,
		const typename mtk::tsqr_tc::detail::get_type<compute_mode>::type t
		) {
	if constexpr (compute_mode == mtk::tsqr_tc::compute_mode::fp32_hmma_cor) {
		compute_w_fp32_hmma_cor<smem_m, smem_n, smem_ldm>(smem_w_ptr, smem_reduction_ptr, smem_y_ptr, smem_Y_ptr, smem_W_ptr, t);
	}
}

// This function computes `A <- (I - W * Y^T)A`.
// Restrictions:
// - smem_m == block_size
// - smem_n == DIM_BLOCK_N
template <unsigned smem_m, unsigned smem_n, unsigned smem_ldm>
__device__ void update_a_fp32_hmma_cor(
		float* const smem_A_ptr,
		float* const smem_YtA_ptr,
		const float* const smem_W_ptr,
		const float* const smem_Y_ptr
		) {
	constexpr unsigned num_col_block = warp_size / smem_n;
	const float cor_scale = 1024.0f;

	{
		nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, smem_n, smem_n, smem_n, half, nvcuda::wmma::row_major> frag_Yt[num_col_block], frag_d_Yt[num_col_block];
		nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, smem_n, smem_n, smem_n, half, nvcuda::wmma::col_major> frag_A[num_col_block], frag_d_A[num_col_block];
		// Load Yt
		mtk::wmma::foreach<decltype(frag_Yt[0])>([&](const unsigned frag_index_list[], const unsigned frag_index_count, const unsigned mem_index) {
					const auto offset = (mem_index / smem_n) * smem_ldm;
					const auto row = mem_index % smem_n + (threadIdx.x & 0xffffffe0u);
					for (unsigned k = 0; k < num_col_block; k++) {
						const auto r = row + k * smem_n;
						const auto v = smem_Y_ptr[offset + r];
						const auto hv = cutf::type::cast<half>(v);
						const auto dhv = cutf::type::cast<half>((v - cutf::type::cast<float>(hv)) * cor_scale);
						for (unsigned i = 0; i < frag_index_count; i++) {
						const unsigned frag_index = frag_index_list[i];
							frag_Yt[k].x[frag_index] = hv;
							frag_d_Yt[k].x[frag_index] = dhv;
						}
					}
				});
		// Load A
		mtk::wmma::foreach<decltype(frag_A[0])>([&](const unsigned frag_index_list[], const unsigned frag_index_count, const unsigned mem_index) {
					const auto offset = (mem_index / smem_n) * smem_ldm;
					const auto row = mem_index % smem_n + (threadIdx.x & 0xffffffe0u);
					for (unsigned k = 0; k < num_col_block; k++) {
						const auto r = row + k * smem_n;
						const auto v = smem_A_ptr[offset + r];
						const auto hv = cutf::type::cast<half>(v);
						const auto dhv = cutf::type::cast<half>((v - cutf::type::cast<float>(hv)) * cor_scale);
						for (unsigned i = 0; i < frag_index_count; i++) {
						const unsigned frag_index = frag_index_list[i];
							frag_A[k].x[frag_index] = hv;
							frag_d_A[k].x[frag_index] = dhv;
						}
					}
				});

		nvcuda::wmma::fragment<nvcuda::wmma::accumulator, smem_n, smem_n, smem_n, float> frag_tmp, frag_d_tmp;
		mtk::wmma::fill_zero(frag_tmp);
		mtk::wmma::fill_zero(frag_d_tmp);
		for (unsigned k = 0; k < num_col_block; k++) {
			// Compute (Yt * A)
			nvcuda::wmma::mma_sync(frag_tmp[k]  , frag_Yt[k]  , frag_A[k]  , frag_tmp  );
			nvcuda::wmma::mma_sync(frag_d_tmp[k], frag_d_Yt[k], frag_A[k]  , frag_d_tmp);
			nvcuda::wmma::mma_sync(frag_d_tmp[k], frag_Yt[k]  , frag_d_A[k], frag_d_tmp);
		}

		for (unsigned i = 0; i < frag_tmp.num_elements; i++) {
			frag_tmp.x[i] += frag_d_tmp.x[i] / cor_scale;
		}

		nvcuda::wmma::store_matrix_sync(smem_YtA_ptr + smem_n * smem_n * (threadIdx.x >> 5), frag_tmp, smem_n, nvcuda::wmma::mem_col_major);
	}

	__syncthreads();
	accumulate_vectors<smem_m>(smem_YtA_ptr, smem_n * smem_n);
	if (threadIdx.x < smem_n * smem_n) {
		smem_YtA_ptr[threadIdx.x] *= -1.0f;
	}
	__syncthreads();

	// Compute (A = A - W * YtA)
	{
		nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, smem_n, smem_n, smem_n, half, nvcuda::wmma::col_major> frag_YtA, frag_d_YtA;
		nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, smem_n, smem_n, smem_n, half, nvcuda::wmma::col_major> frag_W[num_col_block], frag_d_W[num_col_block];

		// Load Yt
		mtk::wmma::foreach<decltype(frag_W[0])>([&](const unsigned frag_index_list[], const unsigned frag_index_count, const unsigned mem_index) {
					const auto offset = (mem_index / smem_n) * smem_ldm;
					const auto row = mem_index % smem_n + (threadIdx.x & 0xffffffe0u);
					for (unsigned k = 0; k < num_col_block; k++) {
						const auto r = row + k * smem_n;
						const auto v = smem_W_ptr[offset + r];
						const auto hv = cutf::type::cast<half>(v);
						const auto dhv = cutf::type::cast<half>((v - cutf::type::cast<float>(hv)) * cor_scale);
						for (unsigned i = 0; i < frag_index_count; i++) {
						const unsigned frag_index = frag_index_list[i];
							frag_W[k].x[frag_index] = hv;
							frag_d_W[k].x[frag_index] = dhv;
						}
					}
				});
		// Load A
		mtk::wmma::foreach<decltype(frag_YtA)>([&](const unsigned frag_index_list[], const unsigned frag_index_count, const unsigned mem_index) {
					const auto row = mem_index % smem_n + (threadIdx.x & 0xffffffe0u);
					for (unsigned k = 0; k < num_col_block; k++) {
						const auto v = smem_YtA_ptr[mem_index];
						const auto hv = cutf::type::cast<half>(v);
						const auto dhv = cutf::type::cast<half>((v - cutf::type::cast<float>(hv)) * cor_scale);
						for (unsigned i = 0; i < frag_index_count; i++) {
						const unsigned frag_index = frag_index_list[i];
							frag_YtA[k].x[frag_index] = hv;
							frag_d_YtA[k].x[frag_index] = dhv;
						}
					}
				});

		for (unsigned k = 0; k < num_col_block; k++) {
			nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, smem_n, smem_n, smem_n, float> frag_A, frag_d_A;
			nvcuda::wmma::load_matrix_sync(frag_A, smem_A_ptr + (threadIdx.x & 0xffffffe0u), smem_ldm);
			mtk::wmma::fill_zero(frag_d_A);
			// Compute (Yt * A)
			nvcuda::wmma::mma_sync(frag_A  , frag_W[k]  , frag_YtA  , frag_A  );
			nvcuda::wmma::mma_sync(frag_d_A, frag_d_W[k], frag_YtA  , frag_d_A);
			nvcuda::wmma::mma_sync(frag_d_A, frag_W[k]  , frag_d_YtA, frag_d_A);

			for (unsigned i = 0; i < frag_A.num_elements; i++) {
				frag_A.x[i] += frag_d_A.x[i] / cor_scale;
			}

			nvcuda::wmma::store_matrix_sync(smem_A_ptr + (threadIdx.x & 0xffffffe0u), frag_A, smem_n, nvcuda::wmma::mem_col_major);
		}
	}
}

template <mtk::tsqr_tc::compute_mode::type compute_mode, unsigned smem_m, unsigned smem_n, unsigned smem_ldm>
__device__ void update_a(
		typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const smem_A_ptr,
		const typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const smem_YtA_ptr,
		const typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const smem_W_ptr,
		const typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const smem_Y_ptr
		) {
	if constexpr (compute_mode == mtk::tsqr_tc::compute_mode::fp32_hmma_cor) {
		update_a_fp32_hmma_cor<smem_m, smem_n, smem_ldm>(smem_A_ptr, smem_YtA_ptr, smem_W_ptr, smem_Y_ptr);
	}
}

template <mtk::tsqr_tc::compute_mode::type compute_mode>
__device__ void qr_kernel(
		typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const gmem_w_ptr, const std::size_t ldw,
		typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const gmem_y_ptr, const std::size_t ldy,
		typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const gmem_t_ptr,
		typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const gmem_a_ptr, const std::size_t lda,
		const std::size_t m,
		const std::size_t n
		) {
	using T = typename mtk::tsqr_tc::detail::get_type<compute_mode>::type;
	constexpr unsigned DIM_MAX_M = 256;
	constexpr unsigned DIM_BLOCK_N = 16;
	constexpr unsigned block_size = DIM_MAX_M;
	constexpr unsigned num_warps = block_size / warp_size;

	__shared__ T smem_A[DIM_MAX_M * DIM_BLOCK_N];
	__shared__ T smem_W[DIM_MAX_M * DIM_BLOCK_N];
	__shared__ T smem_Y[DIM_MAX_M * DIM_BLOCK_N];
	__shared__ T smem_t[DIM_BLOCK_N];
	__shared__ T smem_YtA[DIM_BLOCK_N * DIM_BLOCK_N * num_warps];
	__shared__ T smem_reduction[DIM_BLOCK_N * num_warps];

	T* const smem_A_ptr = smem_A;
	T* const smem_W_ptr = smem_W;
	T* const smem_Y_ptr = smem_Y;
	T* const smem_t_ptr = smem_t;
	T* const smem_YtA_ptr = smem_Y;
	T* const smem_reduction_ptr = smem_reduction;

	const unsigned num_n_blocks = (n + DIM_BLOCK_N - 1) / DIM_BLOCK_N;
	for (std::size_t n_block = 0; n_block < num_n_blocks; n_block++) {
		fill_zero<block_size, DIM_MAX_M * DIM_BLOCK_N>(smem_W);
		fill_zero<block_size, DIM_MAX_M * DIM_BLOCK_N>(smem_Y);

		const unsigned real_block_n = umin(DIM_BLOCK_N, n - DIM_BLOCK_N * n_block);
		copy_matrix_g2s<block_size, DIM_MAX_M, DIM_BLOCK_N, DIM_MAX_M>(smem_A_ptr, gmem_a_ptr + lda * n_block * DIM_BLOCK_N, lda, m, real_block_n);

		for (unsigned sn = 0; sn < real_block_n; sn++) {
			const auto gn = n_block * DIM_BLOCK_N + sn;

			// Copy y from A
			if (threadIdx.x >= gn) {
				const auto index = DIM_MAX_M * sn + threadIdx.x;
				smem_Y_ptr[index] = smem_A_ptr[index];
			}
			__syncthreads();

			// Compute norm2 of y and update y (y_i <- y_i +- norm(y);
			if (cutf::thread::get_warp_id() == gn / warp_size) {
				const auto norm2 = cutf::type::cast<T>(compute_norm2<float>(smem_Y_ptr + DIM_MAX_M * sn, DIM_MAX_M));
				if (cutf::thread::get_lane_id() == sn) {
					const auto norm = cutf::math::sqrt(norm2);
					const auto y_i = smem_Y_ptr[DIM_MAX_M * sn + threadIdx.x];
					smem_Y_ptr[DIM_MAX_M * sn] = y_i + cutf::math::sign(y_i) * norm;
				}
			}
			__syncthreads();

			// Compute norm2 of y
			// TODO: Compute it from previous norm2
			const auto t = cutf::type::cast<T>(2.0f / compute_norm2<float>(smem_Y_ptr + DIM_MAX_M * sn));
			if (sn == threadIdx.x) {
				smem_t_ptr[sn] = t;
			}
			
			// Compute ytA
			compute_reflection_0<compute_mode, DIM_MAX_M, DIM_BLOCK_N, DIM_MAX_M>(smem_reduction_ptr, smem_Y_ptr + DIM_MAX_M * sn, smem_A_ptr);

			// Compute R
			compute_reflection_1<compute_mode, DIM_MAX_M, DIM_BLOCK_N, DIM_MAX_M>(smem_A_ptr, smem_reduction_ptr, smem_Y_ptr + DIM_MAX_M * sn, t);

			// Compute W
			if (sn == 0) {
				smem_W_ptr[threadIdx.x] = smem_Y_ptr[threadIdx.x] * t;
			} else {
				compute_w<compute_mode, DIM_MAX_M, DIM_BLOCK_N, DIM_MAX_M>(smem_W_ptr + DIM_MAX_M * sn, smem_reduction_ptr, smem_Y_ptr + DIM_MAX_M * sn, smem_Y_ptr, smem_W_ptr, t);
			}
		}
		// Store block A, W, Y, t to global memory
		copy_matrix_s2g<block_size, DIM_MAX_M, DIM_BLOCK_N, DIM_MAX_M>(gmem_a_ptr + lda * n_block * DIM_BLOCK_N, lda, smem_A_ptr, m, real_block_n);
		copy_matrix_s2g<block_size, DIM_MAX_M, DIM_BLOCK_N, DIM_MAX_M>(gmem_w_ptr + ldw * n_block * DIM_BLOCK_N, ldw, smem_W_ptr, m, real_block_n);
		copy_matrix_s2g<block_size, DIM_MAX_M, DIM_BLOCK_N, DIM_MAX_M>(gmem_y_ptr + ldy * n_block * DIM_BLOCK_N, ldy, smem_Y_ptr, m, real_block_n);
		if (threadIdx.x < DIM_BLOCK_N) {
			gmem_t_ptr[n_block * DIM_BLOCK_N + threadIdx.x] = smem_t_ptr[threadIdx.x];
		}

		// Update A
		for (std::size_t sub_n_block = n_block + 1; sub_n_block < num_n_blocks; sub_n_block++) {
			const unsigned real_block_n = umin(DIM_BLOCK_N, n - DIM_BLOCK_N * sub_n_block);
			copy_matrix_g2s<block_size, DIM_MAX_M, DIM_BLOCK_N, DIM_MAX_M>(smem_A_ptr, gmem_a_ptr + lda * sub_n_block * DIM_BLOCK_N, lda, m, real_block_n);
			update_a<compute_mode, DIM_MAX_M, DIM_BLOCK_N, DIM_MAX_M>(smem_A_ptr, smem_YtA_ptr, smem_W_ptr, smem_Y_ptr);
			copy_matrix_s2g<block_size, DIM_MAX_M, DIM_BLOCK_N, DIM_MAX_M>(gmem_a_ptr + lda * sub_n_block * DIM_BLOCK_N, lda, smem_A_ptr, m, real_block_n);
		}
	}
}

template <mtk::tsqr_tc::compute_mode::type compute_mode>
__global__ void qr256x128_kernel(
		typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const gmem_w_ptr, const std::size_t ldw,
		typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const gmem_y_ptr, const std::size_t ldy,
		typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const gmem_t_ptr,
		typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const gmem_a_ptr, const std::size_t lda,
		const std::size_t m,
		const std::size_t n) {
	qr_kernel<compute_mode>(
			gmem_w_ptr, ldw,
			gmem_y_ptr, ldy,
			gmem_t_ptr,
			gmem_a_ptr, lda,
			m, n
			);
}
} // noname namespace

template <mtk::tsqr_tc::compute_mode::type compute_mode>
void mtk::tsqr_tc::qr256x128(
		typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const gmem_w_ptr, const std::size_t ldw,
		typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const gmem_y_ptr, const std::size_t ldy,
		typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const gmem_t_ptr,
		typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const gmem_a_ptr, const std::size_t lda,
		const std::size_t m,
		const std::size_t n) {
	const unsigned block_size = 256;
	qr256x128_kernel<compute_mode><<<1, block_size>>>(
			gmem_w_ptr, ldw,
			gmem_y_ptr, ldy,
			gmem_t_ptr,
			gmem_a_ptr, lda,
			m, n
			);
}

#define QR256X128_INSTANCE(compute_mode) \
template <> void mtk::tsqr_tc::qr256x128<compute_mode>( \
		typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const, const std::size_t, \
		typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const, const std::size_t, \
		typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const, \
		typename mtk::tsqr_tc::detail::get_type<compute_mode>::type* const, const std::size_t, \
		const std::size_t, \
		const std::size_t)

QR256X128_INSTANCE(mtk::tsqr_tc::compute_mode::fp32_hmma_cor);
