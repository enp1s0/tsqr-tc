#include "hip/hip_runtime.h"
#include <cstdint>
#include <cutf/type.hpp>
#include <cutf/math.hpp>
#include <cutf/thread.hpp>

namespace {

constexpr unsigned warp_size = 32u;

// This function fills memory with zero
template <unsigned block_size, unsigned size, class T>
__device__ void fill_zero(T* const ptr) {
	if constexpr (size % block_size == 0) {
		for (unsigned i = 0; i < size; i += block_size) {
			const auto index = i + threadIdx.x;
			ptr[index] = cutf::type::cast<T>(0.0f);
		}
	} else {
		for (unsigned index = threadIdx.x; index < size; index += block_size) {
			ptr[index] = cutf::type::cast<T>(0.0f);
		}
	}
}

// This function copies matrix data from global memory to shared memory
// Ristrictions:
// - smem_m == block_size
template <unsigned block_size, unsigned smem_n, unsigned smem_ld, class SMEM_T, class GMEM_T>
__device__ void copy_matrix_g2s(
		SMEM_T* const smem,
		const GMEM_T* const gmem_ptr, const std::size_t gmem_ld,
		const std::size_t m, const std::size_t n
		) {
	if (m == block_size) {
		unsigned i_n = 0;
		for (; i_n < n; i_n++) {
			const auto v = gmem_ptr[gmem_ld * i_n + threadIdx.x];
			smem[smem_ld * i_n + threadIdx.x] = cutf::type::cast<SMEM_T>(v);
		}
		for (; i_n < smem_n; i_n++) {
			smem[smem_ld * i_n + threadIdx.x] = cutf::type::cast<SMEM_T>(0.0f);
		}
	} else {
		unsigned i_n = 0;
		for (; i_n < n; i_n++) {
			const auto v = cutf::type::cast<GMEM_T>(0.0f);
			if (threadIdx.x < m) {
				v = gmem_ptr[gmem_ld * i_n + threadIdx.x];
			}
			smem[smem_ld * i_n + threadIdx.x] = cutf::type::cast<SMEM_T>(v);
		}
		for (; i_n < smem_n; i_n++) {
			smem[smem_ld * i_n + threadIdx.x] = cutf::type::cast<SMEM_T>(0.0f);
		}
	}
}

// This function computes L2-norm ^2 of a given vector(array).
// Restrictions:
// - size % warp_size == 0
template <class COMPUTE_T, class T>
__device__ COMPUTE_T compute_norm2(const T* const ptr, const unsigned size) {
	auto norm2 = cutf::type::cast<COMPUTE_T>(0.0f);
	for (unsigned i = 0; i < size; i += warp_size) {
		const auto v = cutf::type::cast<COMPUTE_T>(ptr[i + cutf::thread::get_lane_id()]);
		norm2 += v * v;
	}
	for(auto mask = (warp_size >> 1); mask > 0; mask >>= 1) {
		norm2 += __shfl_xor_sync(0xffffffff, norm2, mask);
	}
	return norm2;
}
} // noname namespace
