#include "hip/hip_runtime.h"
#include <cstdint>
#include <cutf/type.hpp>
#include <cutf/math.hpp>
#include <cutf/thread.hpp>

namespace {

constexpr unsigned warp_size = 32u;

// This function fills memory with zero
template <unsigned block_size, unsigned size, class T>
__device__ void fill_zero(T* const ptr) {
	if constexpr (size % block_size == 0) {
		for (unsigned i = 0; i < size; i += block_size) {
			const auto index = i + threadIdx.x;
			ptr[index] = cutf::type::cast<T>(0.0f);
		}
	} else {
		for (unsigned index = threadIdx.x; index < size; index += block_size) {
			ptr[index] = cutf::type::cast<T>(0.0f);
		}
	}
}

// This function copies matrix data from global memory to shared memory
// Ristrictions:
// - smem_m == block_size
template <unsigned block_size, unsigned smem_n, unsigned smem_ld, class SMEM_T, class GMEM_T>
__device__ void copy_matrix_g2s(
		SMEM_T* const smem,
		const GMEM_T* const gmem_ptr, const std::size_t gmem_ld,
		const std::size_t m, const std::size_t n
		) {
	if (m == block_size) {
		unsigned i_n = 0;
		for (; i_n < n; i_n++) {
			const auto v = gmem_ptr[gmem_ld * i_n + threadIdx.x];
			smem[smem_ld * i_n + threadIdx.x] = cutf::type::cast<SMEM_T>(v);
		}
		for (; i_n < smem_n; i_n++) {
			smem[smem_ld * i_n + threadIdx.x] = cutf::type::cast<SMEM_T>(0.0f);
		}
	} else {
		unsigned i_n = 0;
		for (; i_n < n; i_n++) {
			const auto v = cutf::type::cast<GMEM_T>(0.0f);
			if (threadIdx.x < m) {
				v = gmem_ptr[gmem_ld * i_n + threadIdx.x];
			}
			smem[smem_ld * i_n + threadIdx.x] = cutf::type::cast<SMEM_T>(v);
		}
		for (; i_n < smem_n; i_n++) {
			smem[smem_ld * i_n + threadIdx.x] = cutf::type::cast<SMEM_T>(0.0f);
		}
	}
}

// This function computes L2-norm ^2 of a given vector(array).
// Restrictions:
// - size % warp_size == 0
template <class COMPUTE_T, class T>
__device__ COMPUTE_T compute_norm2(const T* const ptr, const unsigned size) {
	auto norm2 = cutf::type::cast<COMPUTE_T>(0.0f);
	for (unsigned i = 0; i < size; i += warp_size) {
		const auto v = cutf::type::cast<COMPUTE_T>(ptr[i + cutf::thread::get_lane_id()]);
		norm2 += v * v;
	}
	for(auto mask = (warp_size >> 1); mask > 0; mask >>= 1) {
		norm2 += __shfl_xor_sync(0xffffffff, norm2, mask);
	}
	return norm2;
}

// This function accumulates vectors on shared memory.
// Restrictions:
// count == block_size / war_size
// output_ptr == inpute_ptr
template <unsigned block_size, class T>
__device__ void accumulate_vectors(T* const smem_vec_ptr, const unsigned vec_len) {
	for (unsigned whole_vec_len = vec_len * block_size / warp_size; whole_vec_len > vec_len; whole_vec_len >>= 1) {
		for (unsigned offset = 0; offset < whole_vec_len / 2; offset += block_size) {
			const auto index = offset + threadIdx.x;
			if (index > vec_len) break;

			smem_vec_ptr[index] += smem_vec_ptr[index + whole_vec_len / 2];
		}
		__syncthreads();
	}
}
} // noname namespace
