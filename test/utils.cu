#include "hip/hip_runtime.h"
#include "utils.hpp"
#include <cutf/memory.hpp>
#include <cutf/type.hpp>

namespace {
template <class DST_T, class SRC_T>
__global__ void convert_matrix_kernel(
		DST_T* const dst_matrix_ptr, const std::size_t ld_dst,
		const SRC_T* const src_matrix_ptr, const std::size_t ld_src,
		const std::size_t m, const std::size_t n
		) {
	const auto tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid >= m * n) return;
	const auto im = tid % m;
	const auto in = tid / m;

	dst_matrix_ptr[im + in * ld_dst] = cutf::type::cast<DST_T>(src_matrix_ptr[im + in * ld_src]);
}
template <class DST_T, class SRC_T>
void convert_matrix(
		DST_T* const dst_matrix_ptr, const std::size_t ld_dst,
		const SRC_T* const src_matrix_ptr, const std::size_t ld_src,
		const std::size_t m, const std::size_t n
		) {
	constexpr std::size_t block_size = 256;
	const auto num_threads = m * n;
	convert_matrix_kernel<DST_T, SRC_T><<<(num_threads + block_size - 1) / block_size, block_size>>>(
			dst_matrix_ptr, ld_dst,
			src_matrix_ptr, ld_src,
			m, n
			);
	CUTF_CHECK_ERROR(hipDeviceSynchronize());
}
}

template <class T>
double mtk::tsqr_tc::test_utils::compute_residual_in_dp(
		const T* const dR_ptr, const std::size_t ld_R,
		const T* const dW_ptr, const std::size_t ld_W,
		const T* const dY_ptr, const std::size_t ld_Y,
		const T* const dA_ptr, const std::size_t ld_A,
		const std::size_t m, const std::size_t n,
		hipblasHandle_t const cublas_handle
		) {
	auto hR_dp_uptr = cutf::memory::get_host_unique_ptr<double>(m * n);
	auto hW_dp_uptr = cutf::memory::get_host_unique_ptr<double>(m * n);
	auto hY_dp_uptr = cutf::memory::get_host_unique_ptr<double>(n * n);
	auto hA_dp_uptr = cutf::memory::get_host_unique_ptr<double>(m * n);
	auto hTMP_dp_uptr = cutf::memory::get_host_unique_ptr<double>(n * n);
	convert_matrix(hR_dp_uptr.get(), m, dR_ptr, ld_R, m, n);
	convert_matrix(hW_dp_uptr.get(), m, dW_ptr, ld_W, m, n);
	convert_matrix(hY_dp_uptr.get(), n, dY_ptr, ld_Y, n, n);
	convert_matrix(hA_dp_uptr.get(), m, dA_ptr, ld_A, m, n);
	CUTF_CHECK_ERROR(hipDeviceSynchronize());

	const auto one = 1.0;
	const auto m_one = -1.0;
	const auto zero = 0.0;
	CUTF_CHECK_ERROR(
			cutf::cublas::gemm(
				cublas_handle,
				HIPBLAS_OP_T, HIPBLAS_OP_N,
				n, n, n,
				&one,
				hY_dp_uptr.get(), n,
				hR_dp_uptr.get(), m,
				&zero,
				hTMP_dp_uptr.get(), n
				)
			);
	CUTF_CHECK_ERROR(
			cutf::cublas::gemm(
				cublas_handle,
				HIPBLAS_OP_N, HIPBLAS_OP_N,
				m, n, n,
				&m_one,
				hW_dp_uptr.get(), m,
				hTMP_dp_uptr.get(), n,
				&one,
				hR_dp_uptr.get(), m
				)
			);
	CUTF_CHECK_ERROR(hipDeviceSynchronize());

	// compute_diff
	double base_norm = 0.0;
	double diff_norm = 0.0;
#pragma omp parallel for reduction(+: base_norm) reduction(+: diff_norm)
	for (std::size_t i = 0; i < m * n; i++) {
		const auto diff = hR_dp_uptr.get()[i] - hA_dp_uptr.get()[i];
		const auto base = hA_dp_uptr.get()[i];

		base_norm += base * base;
		diff_norm += diff * diff;
	}

	return std::sqrt(diff_norm / base_norm);
}

template
double mtk::tsqr_tc::test_utils::compute_residual_in_dp<float>(
		const float* const, const std::size_t,
		const float* const, const std::size_t,
		const float* const, const std::size_t,
		const float* const, const std::size_t,
		const std::size_t, const std::size_t,
		hipblasHandle_t const
		);


template <class T>
double mtk::tsqr_tc::test_utils::compute_orthogonality_in_dp(
		const T* const dW_ptr, const std::size_t ld_W,
		const T* const dY_ptr, const std::size_t ld_Y,
		const std::size_t m, const std::size_t n,
		hipblasHandle_t const cublas_handle
		) {
	auto hW_dp_uptr = cutf::memory::get_host_unique_ptr<double>(m * n);
	auto hY_dp_uptr = cutf::memory::get_host_unique_ptr<double>(n * n);
	auto hQ_dp_uptr = cutf::memory::get_host_unique_ptr<double>(m * n);
	auto hE_dp_uptr = cutf::memory::get_host_unique_ptr<double>(n * n);
	convert_matrix(hW_dp_uptr.get(), m, dW_ptr, ld_W, m, n);
	convert_matrix(hY_dp_uptr.get(), n, dY_ptr, ld_Y, n, n);

	// initialize Q
#pragma omp parallel for
	for (std::size_t i = 0; i < m * n; i++) {
		hQ_dp_uptr.get()[i] = 0.0;
	}
	for (std::size_t i = 0; i < n; i++) {
		hQ_dp_uptr.get()[i * (1 + m)] = 1.0;
	}
	// initialize E
#pragma omp parallel for
	for (std::size_t i = 0; i < n * n; i++) {
		hE_dp_uptr.get()[i] = 0.0;
	}
	for (std::size_t i = 0; i < n; i++) {
		hE_dp_uptr.get()[i * (1 + n)] = 1.0;
	}

	const auto one = 1.0;
	const auto m_one = -1.0;
	CUTF_CHECK_ERROR(
			cutf::cublas::gemm(
				cublas_handle,
				HIPBLAS_OP_N, HIPBLAS_OP_T,
				m, n, n,
				&m_one,
				hW_dp_uptr.get(), m,
				hY_dp_uptr.get(), n,
				&one,
				hQ_dp_uptr.get(), m
				)
			);
	CUTF_CHECK_ERROR(
			cutf::cublas::gemm(
				cublas_handle,
				HIPBLAS_OP_T, HIPBLAS_OP_N,
				n, n, m,
				&m_one,
				hQ_dp_uptr.get(), m,
				hQ_dp_uptr.get(), m,
				&one,
				hE_dp_uptr.get(), n
				)
			);
	CUTF_CHECK_ERROR(hipDeviceSynchronize());

	double diff_norm = 0.0;
#pragma omp parallel for reduction(+: diff_norm)
	for (std::size_t i = 0; i < n * n; i++) {
		const auto diff = hE_dp_uptr.get()[i];
		diff_norm += diff * diff;
	}

	return std::sqrt(diff_norm / n);
}

template
double mtk::tsqr_tc::test_utils::compute_orthogonality_in_dp<float>(
		const float* const, const std::size_t,
		const float* const, const std::size_t,
		const std::size_t, const std::size_t,
		hipblasHandle_t const
		);


template <class T>
double mtk::tsqr_tc::test_utils::compute_orthogonality_in_dp(
		const T* const dQ_ptr, const std::size_t ld_Q,
		const std::size_t m, const std::size_t n,
		hipblasHandle_t const cublas_handle
		) {
	auto hQ_dp_uptr = cutf::memory::get_host_unique_ptr<double>(m * n);
	auto hE_dp_uptr = cutf::memory::get_host_unique_ptr<double>(n * n);
	convert_matrix(hQ_dp_uptr.get(), m, dQ_ptr, ld_Q, m, n);

	// initialize E
#pragma omp parallel for
	for (std::size_t i = 0; i < n * n; i++) {
		hE_dp_uptr.get()[i] = 0.0;
	}
	for (std::size_t i = 0; i < n; i++) {
		hE_dp_uptr.get()[i * (1 + n)] = 1.0;
	}

	const auto one = 1.0;
	const auto m_one = -1.0;
	CUTF_CHECK_ERROR(
			cutf::cublas::gemm(
				cublas_handle,
				HIPBLAS_OP_T, HIPBLAS_OP_N,
				n, n, m,
				&m_one,
				hQ_dp_uptr.get(), m,
				hQ_dp_uptr.get(), m,
				&one,
				hE_dp_uptr.get(), n
				)
			);
	CUTF_CHECK_ERROR(hipDeviceSynchronize());

	double diff_norm = 0.0;
#pragma omp parallel for reduction(+: diff_norm)
	for (std::size_t i = 0; i < n * n; i++) {
		const auto diff = hE_dp_uptr.get()[i];
		diff_norm += diff * diff;
	}

	return std::sqrt(diff_norm / n);
}

template
double mtk::tsqr_tc::test_utils::compute_orthogonality_in_dp<float>(
		const float* const, const std::size_t,
		const std::size_t, const std::size_t,
		hipblasHandle_t const
		);

// CUSOLVER QR
namespace {
template <class T>
__global__ void cut_r_kernel(
		T* const dst, const std::size_t ld_dst,
		const T* const src, const std::size_t ld_src,
		const std::size_t n) {
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

	const auto x = tid / n;
	const auto y = tid % n;

	if(y > x) return;

	dst[ld_dst * x + y] = src[ld_src * x + y];
}

template <class T>
void cut_r(T* const dst, const std::size_t ld_dst,
		const T* const src, const std::size_t ld_src,
		const std::size_t n) {
	constexpr std::size_t block_size = 256;
	cut_r_kernel<T><<<(n * n + block_size - 1) / block_size, block_size>>>(
			dst, ld_dst,
			src, ld_src,
			n
			);
}
}

template <class T>
void mtk::tsqr_tc::test_utils::qr_cublas(
		T* const dQ_ptr, const std::size_t ld_Q,
		T* const dR_ptr, const std::size_t ld_R,
		const T* const dA_ptr, const std::size_t ld_A,
		const std::size_t m, const std::size_t n,
		hipsolverHandle_t const cusolver_handle
		) {
	convert_matrix<T, T>(
			dQ_ptr, ld_Q,
			dA_ptr, ld_A,
			m, n
			);
	auto d_tau = cutf::memory::get_device_unique_ptr<T>(n * n);
	int geqrf_working_memory_size, gqr_working_memory_size;
	CUTF_CHECK_ERROR(cutf::cusolver::dn::geqrf_buffer_size(
				cusolver_handle, m, n,
				dQ_ptr, ld_Q, &geqrf_working_memory_size
				));
	CUTF_CHECK_ERROR(cutf::cusolver::dn::gqr_buffer_size(
				cusolver_handle, m, n, n,
				dQ_ptr, ld_Q, d_tau.get(), &gqr_working_memory_size
				));

	auto d_geqrf_working_memory = cutf::memory::get_device_unique_ptr<T>(geqrf_working_memory_size);
	auto d_gqr_working_memory = cutf::memory::get_device_unique_ptr<T>(gqr_working_memory_size);
	auto d_info = cutf::memory::get_device_unique_ptr<int>(1);

	CUTF_CHECK_ERROR(cutf::cusolver::dn::geqrf(
				cusolver_handle, m, n,
				dQ_ptr, ld_Q, d_tau.get(), d_geqrf_working_memory.get(),
				geqrf_working_memory_size, d_info.get()
				));
	cut_r(
			dR_ptr, ld_R,
			dQ_ptr, ld_A,
			n
			);

	CUTF_CHECK_ERROR(cutf::cusolver::dn::gqr(
				cusolver_handle, m, n, n,
				dQ_ptr, ld_Q,
				d_tau.get(), d_gqr_working_memory.get(), gqr_working_memory_size,
				d_info.get()
				));

}

template
void mtk::tsqr_tc::test_utils::qr_cublas<float>(
		float* const dQ_ptr, const std::size_t ld_Q,
		float* const dR_ptr, const std::size_t ld_R,
		const float* const dA_ptr, const std::size_t ld_A,
		const std::size_t m, const std::size_t n,
		hipsolverHandle_t const cusolver_handle
		);
