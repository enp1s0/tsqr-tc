#include <iostream>
#include <random>
#include <chrono>
#include <tsqr_tc/tsqr.hpp>
#include <cutf/memory.hpp>
#include <cutf/cublas.hpp>
#include <cutf/type.hpp>
#include <cutf/stream.hpp>
#include <cutf/debug/matrix.hpp>

#include "utils.hpp"

constexpr float rand_abs_max = 1.0f;
constexpr unsigned test_count = 16;

//#define MTK_DEBUG_PRINT

template <mtk::tsqr_tc::compute_mode::type compute_mode>
void test_accuracy(const std::size_t m, const std::size_t n, const unsigned test_count) {
	using compute_t = typename mtk::tsqr_tc::detail::get_type<compute_mode>::type;
	auto hA_uptr = cutf::memory::get_host_unique_ptr<compute_t>(m * n);
#ifdef MTK_DEBUG_PRINT
	auto hQ_uptr = cutf::memory::get_host_unique_ptr<compute_t>(m * n);
	auto hR_uptr = cutf::memory::get_host_unique_ptr<compute_t>(n * n);
#endif

	auto dA_uptr = cutf::memory::get_device_unique_ptr<compute_t>(m * n);
	auto dQ_uptr = cutf::memory::get_device_unique_ptr<compute_t>(m * n);
	auto dR_uptr = cutf::memory::get_device_unique_ptr<compute_t>(n * n);

	mtk::tsqr_tc::tsqr_buffer<compute_mode> tsqr_buffer(m, n);
	tsqr_buffer.allocate();

	double orthogonality = 0.;
	double residual = 0.;

	for (unsigned c = 0; c < test_count; c++) {
		// initialize input matrix
		{
			std::mt19937 mt(std::random_device{}());
			std::uniform_real_distribution<float> dist(-rand_abs_max, rand_abs_max);
			for (unsigned i = 0; i < m * n; i++) {
				hA_uptr.get()[i] = cutf::type::cast<compute_t>(dist(mt));
			}
		}
#ifdef MTK_DEBUG_PRINT
		cutf::debug::print::print_numpy_matrix(hA_uptr.get(), m, n, "Input_A");
#endif

		cutf::memory::copy(dA_uptr.get(), hA_uptr.get(), m * n);

		auto cuda_stream_uptr = cutf::stream::get_stream_unique_ptr();

		mtk::tsqr_tc::tsqr(
				dQ_uptr.get(), m,
				dR_uptr.get(), n,
				dA_uptr.get(), m,
				m, n,
				tsqr_buffer,
				*cuda_stream_uptr.get()
				);
		CUTF_CHECK_ERROR(hipDeviceSynchronize());

		auto cublas_handle = cutf::cublas::get_cublas_unique_ptr();

		const auto o = mtk::tsqr_tc::test_utils::compute_orthogonality_in_dp(
				dQ_uptr.get(), m,
				m, n,
				*cublas_handle.get()
				);
		const auto r = mtk::tsqr_tc::test_utils::compute_residual_in_dp(
				dQ_uptr.get(), m,
				dR_uptr.get(), n,
				dA_uptr.get(), m,
				m, n,
				*cublas_handle.get()
				);
		residual += r;
		orthogonality += o;
#ifdef MTK_DEBUG_PRINT
		cutf::memory::copy(hR_uptr.get(), dR_uptr.get(), n * n);
		cutf::memory::copy(hQ_uptr.get(), dQ_uptr.get(), m * n);
		cutf::debug::print::print_numpy_matrix(hR_uptr.get(), n, n, "Output_R");
		cutf::debug::print::print_numpy_matrix(hQ_uptr.get(), m, n, "Outout_Q");
#endif
	}

	residual /= test_count;
	orthogonality /= test_count;

	std::printf("%lu,%lu,%s,%e,%e\n", m, n, mtk::tsqr_tc::test_utils::get_mode_name<compute_mode>(), residual, orthogonality);
	std::fflush(stdout);
}

template <mtk::tsqr_tc::compute_mode::type compute_mode>
void test_performance(const std::size_t m, const std::size_t n, const unsigned test_count) {
	using compute_t = typename mtk::tsqr_tc::detail::get_type<compute_mode>::type;
	auto hA_uptr = cutf::memory::get_host_unique_ptr<compute_t>(m * n);

	auto dA_uptr = cutf::memory::get_device_unique_ptr<compute_t>(m * n);
	auto dQ_uptr = cutf::memory::get_device_unique_ptr<compute_t>(m * n);
	auto dR_uptr = cutf::memory::get_device_unique_ptr<compute_t>(n * n);

	mtk::tsqr_tc::tsqr_buffer<compute_mode> tsqr_buffer(m, n);
	tsqr_buffer.allocate();

	// initialize input matrix
	{
		std::mt19937 mt(std::random_device{}());
		std::uniform_real_distribution<float> dist(-rand_abs_max, rand_abs_max);
		for (unsigned i = 0; i < m * n; i++) {
			hA_uptr.get()[i] = cutf::type::cast<compute_t>(dist(mt));
		}
	}
	cutf::memory::copy(dA_uptr.get(), hA_uptr.get(), m * n);

	auto cuda_stream_uptr = cutf::stream::get_stream_unique_ptr();

	const auto start_clock = std::chrono::high_resolution_clock::now();
	for (unsigned c = 0; c < test_count; c++) {
		mtk::tsqr_tc::tsqr(
				dQ_uptr.get(), m,
				dR_uptr.get(), n,
				dA_uptr.get(), m,
				m, n,
				tsqr_buffer,
				*cuda_stream_uptr.get()
				);
	}
	CUTF_CHECK_ERROR(hipDeviceSynchronize());
	const auto end_clock = std::chrono::high_resolution_clock::now();

	const double elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(end_clock - start_clock).count() * 1e-6 / test_count;

	std::printf("%lu,%lu,%s,%e,%lu\n", m, n, mtk::tsqr_tc::test_utils::get_mode_name<compute_mode>(), elapsed_time, tsqr_buffer.get_buffer_size());
	std::fflush(stdout);
}

int main() {
	std::printf("m,n,mode,time,buffer_size\n");
	for (std::size_t lm = 10; lm < 20; lm++) {
		test_accuracy<mtk::tsqr_tc::compute_mode::fp32_hmma_cor>(1lu << lm, 128, test_count);
	}

	for (std::size_t lm = 10; lm < 20; lm++) {
		test_performance<mtk::tsqr_tc::compute_mode::fp32_hmma_cor>(1lu << lm, 128, test_count);
	}

	for (std::size_t lm = 10; lm < 20; lm++) {
		mtk::tsqr_tc::test_utils::test_performance_cublas<float>(1lu << lm, 128, test_count);
	}
}
